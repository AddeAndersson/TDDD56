#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>

__global__ void add_matrix_cuda(float *a, float *b, float *c) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x) + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}

void add_matrix(float *a, float *b, float *c, int N)
{
	int index;
	
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{
	const int N = 2048;
	const int blocksize = 16;
	const int size = N*N*sizeof(float);

	float *a = new float[N*N];
	float *b = new float[N*N];
	float *res = new float[N*N];
	float *a_dev;
	float *b_dev;
	float *c;

	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	
	//add_matrix(a, b, c, N);
	hipMalloc( (void**)&a_dev, size );
	hipMalloc( (void**)&b_dev, size );
	hipMalloc( (void**)&c, size );

	float milliseconds = 0;
	hipEventRecord(start, 0); // Start timer
	hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
	hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);
	dim3 numOfThreads( blocksize, blocksize);
	dim3 numOfBlocks( N / numOfThreads.x, N / numOfThreads.y );

	add_matrix_cuda<<<numOfBlocks, numOfThreads>>>(a_dev, b_dev, c);
	
	hipDeviceSynchronize();
	hipMemcpy( res, c, size, hipMemcpyDeviceToHost ); 
	hipEventRecord(stop, 0); // End timer
	hipEventSynchronize(stop);

	hipEventElapsedTime(&milliseconds, start, stop);

	hipFree( a_dev );
	hipFree( b_dev );
	hipFree( c );

	// for (int i = 0; i < N; i++)
	// {
	// 	for (int j = 0; j < N; j++)
	// 	{
	// 		printf("%0.2f ", res[i+j*N]);
	// 	}
	// 	printf("\n");
	// }
	printf("Time elapsed: %fms\n", milliseconds);

	delete[] a;
	delete[] b;
	delete[] res;
}
